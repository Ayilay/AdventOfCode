#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string.h>
#include <assert.h>
#include <math.h>

#define NVERBOSE
#define NDEBUG
#include "problem_solver.h"

//--------------------------------------------------------------------------------
//  Private State
//--------------------------------------------------------------------------------

#define MAX 400

int N = 0;
char table[ MAX ][ MAX ] = {0};

unsigned int total = 0;

void printTable( const char table[][ MAX ], int n )
{
    printf( "{\n" );
    for( int r = 0; r < n; r++ ){
        printf("  ");
        for( int c = 0; c < n; c++ ){
            printf( "%c, ", table[r][c] );
        }
        printf( "\n" );
    }
    printf( "}\n" );

}

//--------------------------------------------------------------------------------
//  Kernels
//--------------------------------------------------------------------------------

__device__ char arrayGet( const char* array, const int n, const int r, const int c )
{
    return array[ (r*n) + c ];
}

__global__ void basicKernel()
{
    int x = threadIdx.x;
    int y = blockIdx.x;
    printf( "hi from (%d,%d)\n", x, y );

}

/** Note: input is 2D square matrix of dim 'n'
 */
__global__ void Xcompare_1( const char* input, char* output, const int n )
{
    int centerX = 1;
    int centerY = 1;
    // TODO obtain these from threadId

    // If center is NOT A then we DEFINITELY don't have a X-MAS
    int a  = 'A' - arrayGet( input, n, centerX, centerY );
    if( a ){
        return;
    }

    int m1 = 'M' - arrayGet( input, n, centerX-1, centerY-1 );
    int m2 = 'M' - arrayGet( input, n, centerX-1, centerY+1 );
    int s1 = 'S' - arrayGet( input, n, centerX+1, centerY-1 );
    int s2 = 'S' - arrayGet( input, n, centerX+1, centerY+1 );

    int result = m1 + m2 + s1 + s2;
    // TODO store result
}

//--------------------------------------------------------------------------------
//  API Functions
//--------------------------------------------------------------------------------

void SOLVER_Init( void* arg )
{
}

void SOLVER_ProcessLine( char* line )
{
    rmnewl( line );

    int len = strlen( line );
    assert( len < MAX );

    strncpy( table[N], line, MAX );
    N++;
}


void SOLVER_PrintSolution( void )
{
    char transformed[ MAX ][ MAX ];
    memset( transformed, 0, sizeof( transformed ) );

    basicKernel<<< 1, 1 >>> ();

    printf( "Matches: %u\n", total );
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string.h>
#include <assert.h>
#include <math.h>

#define NVERBOSE
#define NDEBUG
#include "problem_solver.h"

//--------------------------------------------------------------------------------
//  Private State
//--------------------------------------------------------------------------------

#define MAX 400

int N = 0;
char table[ MAX ][ MAX ] = {0};

unsigned int total = 0;

void printTable( const char table[][ MAX ], int n )
{
    printf( "{\n" );
    for( int r = 0; r < n; r++ ){
        printf("  ");
        for( int c = 0; c < n; c++ ){
            printf( "%c, ", table[r][c] );
        }
        printf( "\n" );
    }
    printf( "}\n" );

}

//--------------------------------------------------------------------------------
//  Kernels
//--------------------------------------------------------------------------------

__device__ 
char arrayGet( const char* array, const int n, const int r, const int c )
{
    return array[ (r*n) + c ];
}

__device__ 
void arraySet( char* array, const int n, const int r, const int c, char val )
{
    array[ (r*n) + c ] = val;
}

const char kernel_1[] = 
    "M.M"
    ".A."
    "S.S";

const char kernel_2[] = 
    "S.S"
    ".A."
    "M.M";

const char kernel_3[] = 
    "M.S"
    ".A."
    "M.S";

const char kernel_4[] = 
    "S.M"
    ".A."
    "S.M";


/** Note: input is 2D square matrix of dim 'n'
 */
__global__ void Xcompare(
                    const char* input,
                    const char* kernel,
                    char* output,
                    const int n
)
{
    // 2D Convolution goes from 1 to n-1
    int centerX = threadIdx.x + (blockIdx.x * blockDim.x) + 1;
    int centerY = threadIdx.y + (blockIdx.y * blockDim.y) + 1;

    // If center is NOT A then we DEFINITELY don't have a X-MAS
    int a  = 'A' - arrayGet( input, n, centerX, centerY );
    if( a ){
        return;
    }

    // Compare the 4 'corners' of the input to those of the kernel
    int i1 = arrayGet( input, n, centerX-1, centerY-1 );
    int i2 = arrayGet( input, n, centerX-1, centerY+1 );
    int i3 = arrayGet( input, n, centerX+1, centerY-1 );
    int i4 = arrayGet( input, n, centerX+1, centerY+1 );

    int k1 = arrayGet( kernel, 3, 0, 0 );
    int k2 = arrayGet( kernel, 3, 0, 2 );
    int k3 = arrayGet( kernel, 3, 2, 0 );
    int k4 = arrayGet( kernel, 3, 2, 2 );

    int m1 = k1 - i1;
    int m2 = k2 - i2;
    int s1 = k3 - i3;
    int s2 = k4 - i4;

    int result = m1 + m2 + s1 + s2;
    //printf( "Result from (%3d,%3d): %d\n", centerX, centerY, result );
    //printf( "Result from (%3d,%3d): compare %c to %c\n", centerX, centerY, k1, i1 );
    //printf( "Result from (%3d,%3d): compare %c to %c\n", centerX, centerY, k2, i2 );
    //printf( "Result from (%3d,%3d): compare %c to %c\n", centerX, centerY, k3, i3 );
    //printf( "Result from (%3d,%3d): compare %c to %c\n", centerX, centerY, k4, i4 );

    if( result == 0 )
        arraySet( output, n, centerX, centerY, result );
}

//--------------------------------------------------------------------------------
//  API Functions
//--------------------------------------------------------------------------------

void SOLVER_Init( void* arg )
{
}

void SOLVER_ProcessLine( char* line )
{
    rmnewl( line );

    int len = strlen( line );
    assert( len < MAX );

    strncpy( table[N], line, MAX );
    N++;
}

int countMatches( const char* table, int n )
{
    assert( n > 2 );

    int match = 0;

    for( int r = 1; r < n-1; r++ ){
        for( int c = 1; c < n-1; c++ ){ 
            if( table[ (r*n) + c ] == 0 ){
                match++;
            }
        }
    }

    return match;
}

void printTable2( const char* table, int n )
{
    for( int r = 0; r < n; r++ ){
        for( int c = 0; c < n; c++ ){ 
            int elem = table[ (r*n) + c ];
            printf( "%3d ", elem );
        }
        printf("\n");
    }
}

void SOLVER_PrintSolution( void )
{
    char transformed[ MAX ][ MAX ];
    memset( transformed, 0, sizeof( transformed ) );

    char outputs[N*N];

    char* d_inputs;
    char* d_outputs;
    char* d_kernel;
    hipError_t err;

    err = hipMalloc( &d_kernel, 10);
    assert( err == hipSuccess );

    err = hipMalloc( &d_inputs, N * N + 1 );
    assert( err == hipSuccess );

    err = hipMalloc( &d_outputs, N * N );
    hipMemset( d_outputs, 1, N * N );
    assert( err == hipSuccess );

    // Convert standard 2D array to linear array into device memory
    hipMemset( d_inputs, 0, N*N+1 );
    for( int r = 0; r < N; r++){
        int row = r*N;
        hipMemcpy( &d_inputs[row], &table[r][0], N, hipMemcpyHostToDevice );
    }


    assert( N > 2 );
    dim3 problemDim( N-2, N-2 );

    #define LAUNCH( K ) \
        hipMemcpy( d_kernel, kernel_##K, 9, hipMemcpyHostToDevice );  \
        Xcompare<<< problemDim, 1 >>> (d_inputs, d_kernel, d_outputs, N );  \
        hipDeviceSynchronize();

    LAUNCH( 1 )
    LAUNCH( 2 )
    LAUNCH( 3 )
    LAUNCH( 4 )


    hipMemcpy( outputs, d_outputs, N*N, hipMemcpyDeviceToHost );
    total = countMatches( outputs, N );

    printf( "ouitput: \n");
    printTable2( outputs, N );

    printf( "Matches: %u\n", total );

}
